
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */



///////////////////////////////////////////////////////////////////////////////
// On G80-class hardware 24-bit multiplication takes 4 clocks per warp
// (the same as for floating point  multiplication and addition),
// whereas full 32-bit multiplication takes 16 clocks per warp.
// So if integer multiplication operands are  guaranteed to fit into 24 bits
// (always lie withtin [-8M, 8M - 1] range in signed case),
// explicit 24-bit multiplication is preferred for performance.
///////////////////////////////////////////////////////////////////////////////
#define IMUL(a, b) __mul24(a, b)



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
// Parameters restrictions:
// 1) ElementN is strongly preferred to be a multiple of warp size to
//    meet alignment constraints of memory coalescing.
// 2) ACCUM_N must be a power of two.
///////////////////////////////////////////////////////////////////////////////
#define ACCUM_N 1024
__global__ void scalarProdGPU(
    float *d_C,
    float *d_A,
    float *d_B,
    int vectorN,
    int elementN
){
    //Accumulators cache
    __shared__ float accumResult[ACCUM_N];

    ////////////////////////////////////////////////////////////////////////////
    // Cycle through every pair of vectors,
    // taking into account that vector counts can be different
    // from total number of thread blocks
    ////////////////////////////////////////////////////////////////////////////
    for(int vec = blockIdx.x; vec < vectorN; vec += gridDim.x){
        int vectorBase = IMUL(elementN, vec);
        int vectorEnd  = vectorBase + elementN;

        ////////////////////////////////////////////////////////////////////////
        // Each accumulator cycles through vectors with
        // stride equal to number of total number of accumulators ACCUM_N
        // At this stage ACCUM_N is only preferred be a multiple of warp size
        // to meet memory coalescing alignment constraints.
        ////////////////////////////////////////////////////////////////////////
        for(int iAccum = threadIdx.x; iAccum < ACCUM_N; iAccum += blockDim.x){
            float sum = 0;

            for(int pos = vectorBase + iAccum; pos < vectorEnd; pos += ACCUM_N)
                sum += d_A[pos] * d_B[pos];

            accumResult[iAccum] = sum;
        }

        ////////////////////////////////////////////////////////////////////////
        // Perform tree-like reduction of accumulators' results.
        // ACCUM_N has to be power of two at this stage
        ////////////////////////////////////////////////////////////////////////
        for(int stride = ACCUM_N / 2; stride > 0; stride >>= 1){
            __syncthreads();
            for(int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x)
                accumResult[iAccum] += accumResult[stride + iAccum];
        }

        if(threadIdx.x == 0) d_C[vec] = accumResult[0];
    }
}
